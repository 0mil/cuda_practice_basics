// CUDA-02: Vector Addition

// Implemented parallel addition of two float vectors.
// Used thread index = `threadIdx.x + blockDim.x * blockIdx.x`
// Grid size: `ceil(N / threadsPerBlock)`
// Verified result for first 10 elements in result.log

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 100;
    size_t size = N * sizeof(float);

    // Host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Init input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // Device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy input to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print some results
    for (int i = 0; i < 10; i++) {
        printf("C[%d] = %.1f\n", i, h_C[i]);
    }

    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
